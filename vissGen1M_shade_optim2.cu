#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <ctime>
#include <string>
#include <cmath>
#include <omp.h>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "error.cuh"
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <vector>
#include <chrono>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>

#define _USE_MATH_DEFINES
#define EXP 0.0000000000

using namespace std;
using Complex = thrust::complex<float>;

// complexExp 函数的实现
__device__ thrust::complex<float> complexExp(const Complex &d) {
    float realPart = exp(d.real()) * cos(d.imag());
    float imagPart = exp(d.real()) * sin(d.imag());
    return thrust::complex<float>(realPart, imagPart);
}
// complexAbs 函数的实现
__device__ thrust::complex<float> ComplexAbs(const Complex &d) {
    // 复数的模定义为 sqrt(real^2 + imag^2)
    return thrust::complex<float>(sqrt(d.real() * d.real() + d.imag() * d.imag()));
}

struct timeval start, finish;
float total_time;

string address = "./frequency_1M/";
string F_address = "./F_recon_1M/";
string para;
string duration = "frequency1M";  // 第几个周期的uvw
string sufix = ".txt";

// 1 M
const int uvw_presize = 4000000;

// 定义常量
#define BLOCK_SIZE 128                     // 线程块大小
#define SHARED_MEM_SIZE BLOCK_SIZE         // 共享内存大小
#define MAX_THREADS_PER_BLOCK 1024        // GPU每个块的最大线程数


// 计算特定 q 值的索引有几个
__global__ void computeLocCount(
    const float* __restrict__ NX,      // 添加 const 和 __restrict__
    int* __restrict__ countLoc,
    const int lmnC_index,
    const int NX_index)
{
    const int q = blockIdx.x * blockDim.x + threadIdx.x;
    if (q >= lmnC_index) return;

    // 使用局部计数器
    int local_count = 0;
    
    // 主循环展开
    #pragma unroll 4
    for (int i = 0; i < NX_index; i++) {
        // 预取数据到寄存器
        const float nx_val = NX[i] - 1;
        // 使用直接比较代替条件语句
        local_count += (nx_val == q);
    }
    
    // 存储最终结果
    countLoc[q] = local_count;
}


// 把每个 q 值对应的索引保存下来
__global__ void computeLocViss(
    const float* __restrict__ NX,
    int* __restrict__ NXq,
    const int* __restrict__ countLoc,
    const int lmnC_index,
    const int NX_index)
{
    const int q = blockIdx.x * blockDim.x + threadIdx.x;
    if (q >= lmnC_index) return;

    // 预计算起始索引
    int start_idx = 0;
    if (q > 0) {
        #pragma unroll 4
        for (int i = 0; i < q; i++) {
            start_idx += countLoc[i];
        }
    }

    // 使用局部索引计数器
    int local_idx = start_idx;
    
    // 主循环展开
    #pragma unroll 4
    for (int i = 0; i < NX_index; i++) {
        // 预取数据到寄存器
        const float nx_val = NX[i] - 1;
        // 使用条件赋值代替if语句
        bool match = (nx_val == q);
        if (match) {
            NXq[local_idx++] = i;
        }
    }
}


// 定义计算可见度核函数
__global__ void visscal(
    const int uvws_index,
    const int lmnC_index,
    const int res,
    const float* __restrict__ FF,
    Complex* __restrict__ viss, 
    const float* __restrict__ u,
    const float* __restrict__ v,
    const float* __restrict__ w,
    const float* __restrict__ l,
    const float* __restrict__ m,
    const float* __restrict__ n,
    const int* __restrict__ shadeM1,
    const int* __restrict__ shadeM2,
    const int* __restrict__ shadeM3,
    const int* __restrict__ shadeM4,
    const int* __restrict__ NXq,
    const int* __restrict__ countLoc,
    const Complex I1,
    const Complex CPI,
    const Complex zero,
    const Complex two, 
    const float dl,
    const float dm,
    const float dn)
{
    const int uvws_ = blockIdx.x * blockDim.x + threadIdx.x;
    if (uvws_ >= uvws_index) return;

    // 预加载频繁使用的数据到寄存器
    const float u_val = u[uvws_] / dl;
    const float v_val = v[uvws_] / dm;
    const float w_val = w[uvws_] / dn;
    
    // 预加载遮挡相关的数据
    const int shade_m1 = shadeM1[uvws_];
    const int shade_m2 = shadeM2[uvws_];
    const int shade_m3 = shadeM3[uvws_];
    const int shade_m4 = shadeM4[uvws_];

    // 初始化累加器
    Complex acc = zero;
    int start_idx = 0;

    for (int lmnC_ = 0; lmnC_ < lmnC_index; ++lmnC_) {
        const int current_count = countLoc[lmnC_];
        float sumReal = 0.0f;

        for (int con = 0; con < current_count; ++con) {
            const int locViss = NXq[con + start_idx];
            float addFF = FF[locViss];

            // 优化遮挡检查
            for (int lo = 0; lo >= shade_m3 && lo <= shade_m4; ++lo) {
                if (locViss >= lo*res+shade_m1+1 && locViss < lo*res+shade_m2) {
                    addFF = 240;
                }
            }
            sumReal += addFF;
        }
        
        start_idx += current_count;
        const float C_tmp = sumReal / current_count;

        // 计算相位和复指数
        const float phase = u_val * l[lmnC_] + v_val * m[lmnC_] + w_val * (n[lmnC_] - 1.0f);
        const Complex exp_val = complexExp((zero - I1) * two * CPI * Complex(phase, 0.0f));
        acc += Complex(C_tmp, 0.0f) * exp_val;    
    }

    // 计算最终结果
    const Complex final_exp = complexExp((zero - I1) * two * CPI * Complex(w_val, 0.0f));
    viss[uvws_] = acc * final_exp;
}

void launch_visscal(
    const int uvws_index,
    const int lmnC_index,
    const int res,
    Complex* d_viss,
    const float* d_FF,
    const float* d_u,
    const float* d_v,
    const float* d_w,
    const float* d_l,
    const float* d_m,
    const float* d_n,
    const int* d_shadeM1,
    const int* d_shadeM2,
    const int* d_shadeM3,
    const int* d_shadeM4,
    const int* d_NXq,
    const int* d_countLoc,
    const Complex I1,
    const Complex CPI,
    const Complex zero,
    const Complex two,
    const float dl,
    const float dm,
    const float dn)
{
    // 计算网格和块的大小
    int threadsPerBlock;
    int minGridSize; // 最小网格大小
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &threadsPerBlock, visscal, 0, 0);
    int blocksPerGrid = floor(uvws_index + threadsPerBlock - 1) / threadsPerBlock;

    // 创建CUDA流
    const int numStreams = 4;  // 使用4个流
    const int itemsPerStream = uvws_index / numStreams;  // 计算每个流处理的数据量
    
    // 创建流数组
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; i++) {
        hipStreamCreate(&streams[i]);
    }

    // 启动多个流
    for (int i = 0; i < numStreams; i++) {
        const int streamStart = i * itemsPerStream;
        const int streamSize = (i == numStreams-1) ? uvws_index - streamStart : itemsPerStream;
        
        if (streamSize <= 0) break;

        const int streamBlocks = (streamSize + threadsPerBlock - 1) / threadsPerBlock;

        // 启动核函数
        visscal<<<streamBlocks, threadsPerBlock, 0, streams[i]>>>(
            streamSize, lmnC_index, res,
            d_FF, 
            d_viss+streamStart,
            d_u+streamStart, 
            d_v+streamStart, 
            d_w+streamStart,
            d_l, 
            d_m, 
            d_n,
            d_shadeM1+streamStart, 
            d_shadeM2+streamStart, 
            d_shadeM3+streamStart, 
            d_shadeM4+streamStart,
            d_NXq, 
            d_countLoc,
            I1, CPI, zero, two,
            dl, dm, dn
        );
    }

    // 等待所有流完成（如果需要）
    for (int i = 0; i < numStreams; i++) {
        hipStreamSynchronize(streams[i]);
    }

    // 清理流
    for (int i = 0; i < numStreams; i++) {
        hipStreamDestroy(streams[i]);
    }
}


// 定义图像反演核函数  验证正确
__global__ void imagerecon(
    const int uvw_index,
    const int lmnC_index,
    const int res,
    Complex* __restrict__ F,                    
    const Complex* __restrict__ viss,           
    const float* __restrict__ u,
    const float* __restrict__ v,
    const float* __restrict__ w,
    const float* __restrict__ l,
    const float* __restrict__ m,
    const float* __restrict__ n,
    const float* __restrict__ uvwFrequencyMap,
    const float* __restrict__ thetaP0,
    const float* __restrict__ phiP0,
    const float* __restrict__ dtheta,
    const float* __restrict__ dphi,
    const Complex I1,                    
    const Complex CPI,
    const Complex zero,
    const Complex two,
    const float dl,
    const float dm,
    const float dn)
{
    // 声明共享内存
    __shared__ float s_u[SHARED_MEM_SIZE];
    __shared__ float s_v[SHARED_MEM_SIZE];
    __shared__ float s_w[SHARED_MEM_SIZE];
    __shared__ float s_uvwFreq[SHARED_MEM_SIZE];
    __shared__ Complex s_viss[SHARED_MEM_SIZE];
    __shared__ float s_thetaP0[SHARED_MEM_SIZE];
    __shared__ float s_phiP0[SHARED_MEM_SIZE];
    __shared__ float s_dtheta[SHARED_MEM_SIZE];
    __shared__ float s_dphi[SHARED_MEM_SIZE];

    const int lmnC_ = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;
    if (lmnC_ >= lmnC_index) return;

    // 预计算常量
    const Complex amount(uvw_index, 0.0f);
    const float inv_dl = 1.0f / dl;
    const float inv_dm = 1.0f / dm;
    const float inv_dn = 1.0f / dn;
    const float l_val = l[lmnC_] * inv_dl;
    const float m_val = m[lmnC_] * inv_dm;
    const float n_val = n[lmnC_] * inv_dn;

    // 预计算 phiP 和 thetaP
    const float phiP = floorf(lmnC_ / res);
    const float thetaP = lmnC_ - phiP * res;

    // 使用复数累加器
    Complex acc = zero;

    // 使用共享内存分块处理数据
    for (int base = 0; base < uvw_index; base += SHARED_MEM_SIZE) {
        const int current_chunk_size = min(SHARED_MEM_SIZE, uvw_index - base);
        
        // 协作加载数据到共享内存
        for (int i = tid; i < current_chunk_size; i += blockDim.x) {
            const int global_idx = base + i;
            s_u[i] = u[global_idx];
            s_v[i] = v[global_idx];
            s_w[i] = w[global_idx];
            s_uvwFreq[i] = uvwFrequencyMap[global_idx];
            s_viss[i] = viss[global_idx];
            s_thetaP0[i] = thetaP0[global_idx];
            s_phiP0[i] = phiP0[global_idx];
            s_dtheta[i] = dtheta[global_idx];
            s_dphi[i] = dphi[global_idx];
        }
        
        // 确保所有线程完成数据加载
        __syncthreads();

        // 处理当前块中的数据
        #pragma unroll 8
        for (int i = 0; i < current_chunk_size; ++i) {
            // 检查条件
            bool skip_calculation = (fabs(s_thetaP0[i] - thetaP) < s_dtheta[i] && 
                                  fabs(s_phiP0[i] - phiP) < s_dphi[i]);

            if (!skip_calculation) {
                // 计算相位
                const float phase = s_u[i] * l_val + s_v[i] * m_val + s_w[i] * n_val;
                // 计算复指数
                const Complex exp_val = complexExp(I1 * two * CPI * Complex(phase, 0.0f));
                // 累加结果
                acc += s_uvwFreq[i] * s_viss[i] * exp_val;
            }
        }

        // 同步后再处理下一块数据
        __syncthreads();
    }

    // 归一化并存储结果
    F[lmnC_] = acc / amount;
}

// 启动函数
void launch_imagerecon(
    const int uvw_index,
    const int lmnC_index,
    const int res,
    Complex* d_F,
    Complex* d_viss,
    float* d_u,
    float* d_v,
    float* d_w,
    float* d_l,
    float* d_m,
    float* d_n,
    float* d_uvwFrequencyMap,
    float* d_thetaP0,
    float* d_phiP0,
    float* d_dtheta,
    float* d_dphi,
    const Complex I1,
    const Complex CPI,
    const Complex zero,
    const Complex two,
    const float dl,
    const float dm,
    const float dn)
{
    // 计算网格和块的大小
    int threadsPerBlock;
    int minGridSize; // 最小网格大小
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &threadsPerBlock, visscal, 0, 0);
    int blocksPerGrid = floor(lmnC_index + threadsPerBlock - 1) / threadsPerBlock;

    // 计算共享内存大小
    const size_t sharedMemSize = SHARED_MEM_SIZE * (
        sizeof(float) * 8 +    // s_u, s_v, s_w, s_uvwFreq, s_thetaP0, s_phiP0, s_dtheta, s_dphi
        sizeof(Complex)        // s_viss
    );

    // 设置流的数量
    const int numStreams = 4;  // 可以根据需要调整
    const int itemPerStream = (lmnC_index + numStreams - 1) / numStreams;

    // 创建流数组
    hipStream_t streams[numStreams];
    for(int i = 0; i < numStreams; i++) {
        hipStreamCreate(&streams[i]);
    }

    // 设置缓存配置
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(imagerecon), hipFuncCachePreferShared);

    // 启动多个流
    for(int i = 0; i < numStreams; i++) {
        const int streamStart = i * itemPerStream;
        const int streamSize = (i == numStreams-1) ? lmnC_index-streamStart : itemPerStream;

        if(streamSize <= 0) break;

        const int streamBlocks = (streamSize + threadsPerBlock - 1) / threadsPerBlock;

        // 启动核函数，只对 l, m, n 相关的数组进行偏移
        imagerecon<<<streamBlocks, threadsPerBlock, sharedMemSize, streams[i]>>>(
            uvw_index,
            streamSize,        // 这个流处理的 lmnC 数量
            res,
            d_F + streamStart, 
            d_viss,           
            d_u,              
            d_v,              
            d_w,              
            d_l + streamStart, // 只处理部分 l
            d_m + streamStart, // 只处理部分 m
            d_n + streamStart, // 只处理部分 n
            d_uvwFrequencyMap,
            d_thetaP0,
            d_phiP0,
            d_dtheta,
            d_dphi,
            I1, CPI, zero, two,
            dl, dm, dn
        );
    }

    // 等待所有流完成（如果需要）
    for (int i = 0; i < numStreams; i++) {
        hipStreamSynchronize(streams[i]);
    }

    // 清理流
    for (int i = 0; i < numStreams; i++) {
        hipStreamDestroy(streams[i]);
    }
}


int vissGen(int id, int RES, int start_period) 
{   
    cout << "res: " << RES << endl;
    int days = 1;  // 一共有多少个周期  15月 * 30天 / 14天/周期
    cout << "periods: " << days << endl;
    Complex I1(0.0, 1.0);
    float dl = 2 * RES / (RES - 1);
    float dm = 2 * RES / (RES - 1);
    float dn = 2 * RES / (RES - 1);
    Complex zero(0.0, 0.0);
    Complex two(2.0, 0.0);
    Complex CPI(M_PI, 0.0);

    gettimeofday(&start, NULL);
    int nDevices = 1;
    // 设置节点数量（gpu显卡数量）
    CHECK(hipGetDeviceCount(&nDevices));
    // 设置并行区中的线程数
    omp_set_num_threads(nDevices);
    cout << "devices: " << nDevices << endl;

    // 加载存储 l m n C nt的文件（对于不同的frequency不一样，只与frequency有关）
    string para, address_l, address_m, address_n, address_C, address_NX, address_FF;
    ifstream lFile, mFile, nFile, cFile, NXFile, FFFile;
    para = "l";
    address_l = address + para + sufix;
    lFile.open(address_l);
    cout << "address_l: " << address_l << endl;
    para = "m";
    address_m = address + para + sufix;
    mFile.open(address_m);
    cout << "address_m: " << address_m << endl;
    para = "n";
    address_n = address + para + sufix;
    nFile.open(address_n);
    cout << "address_n: " << address_n << endl;
    para = "C";
    address_C = address + para + sufix;
    cFile.open(address_C);
    cout << "address_C: " << address_C << endl;
    para = "NX";
    address_NX = address + para + sufix;
    NXFile.open(address_NX);
    cout << "address_NX: " << address_NX << endl;
    para = "FF";
    address_FF = address + para + sufix;
    FFFile.open(address_FF);
    cout << "address_FF: " << address_FF << endl;
    if (!lFile.is_open() || !mFile.is_open() || !nFile.is_open() || !cFile.is_open() || !NXFile.is_open() ||!FFFile.is_open()) {
        std::cerr << "无法打开一个或多个文件：" << std::endl;
        if (!lFile.is_open()) std::cerr << "无法打开文件: " << address_l << std::endl;
        if (!mFile.is_open()) std::cerr << "无法打开文件: " << address_m << std::endl;
        if (!nFile.is_open()) std::cerr << "无法打开文件: " << address_n << std::endl;
        if (!cFile.is_open()) std::cerr << "无法打开文件: " << address_C << std::endl;
        if (!NXFile.is_open()) std::cerr << "无法打开文件: " << address_NX << std::endl;
        if (!FFFile.is_open()) std::cerr << "无法打开文件: " << address_FF << std::endl;
        return -1; 
    }
    int lmnC_index = 0;
    int NX_index = 0;
    lFile >> lmnC_index;  // 读取l的第一行的行数
    FFFile >> NX_index;  // 读取FF的第一行的行数
    cout << "lmnC index: " << lmnC_index << endl;
    cout << "NX index: " << NX_index << endl;

    std::vector<float> cl(lmnC_index), cm(lmnC_index), cn(lmnC_index), cc(lmnC_index);
    std::vector<float> cNX(NX_index), cFF(NX_index);
    for (int i = 0; i < lmnC_index && lFile.good() && mFile.good() && nFile.good() && cFile.good(); ++i) {
        lFile >> cl[i];
        mFile >> cm[i];
        nFile >> cn[i];
        cFile >> cc[i];
    }
    for (int i = 0; i < NX_index && NXFile.good() && FFFile.good(); ++i) {
        NXFile >> cNX[i];
        FFFile >> cFF[i];
    }
    lFile.close();
    mFile.close();
    nFile.close();
    cFile.close();
    NXFile.close();
    FFFile.close();

    // 导入uvw坐标的出现频次，txt文件的每一行每个坐标的频次
    auto uvwMapStart = std::chrono::high_resolution_clock::now();
    // 创建map存储
    std::unordered_map<std::string, float> cUVWFrequencyMap;
    string uvwmap_address = address + "uvwMap130.txt";
    std::ifstream uvwMapFile(uvwmap_address);
    if (uvwMapFile.is_open()) {
        // 读取第一行获取总行数
        string firstLine;
        std::getline(uvwMapFile, firstLine);
        int totalLines = std::stoi(firstLine);
        cout << "uvwMap totalLines: " << totalLines << endl;
        // 预分配内存
        cUVWFrequencyMap.reserve(totalLines);
        // 每一行的格式： -23 -288 -166 4
        string line;
        while (std::getline(uvwMapFile, line)) {
            std::istringstream iss(line);
            int u_point, v_point, w_point;
            int uvw_frequency;
            if (iss >> u_point >> v_point >> w_point >> uvw_frequency) {
                std::string key = std::to_string(u_point) + "_" + std::to_string(v_point) + "_" + std::to_string(w_point);
                cUVWFrequencyMap[key] = uvw_frequency;
            } else {
                cout << "Failed to parse line: " << line << endl; // 解析失败时的调试信息
            }
        }
        uvwMapFile.close();
    }
    // 打印测试确保是正确的
    int count = 0;
    int numElementsToPrint = 6; // 设定要打印的元素数量
    for (const auto& pair : cUVWFrequencyMap) {
        std::cout << pair.first << ": " << pair.second << std::endl;
        if (++count == numElementsToPrint) {
            break;
        }
    }
    cout << "Transfer uvw Frequency Success! " << endl;
    auto uvwMapFinish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> uvwMapElapsed = uvwMapFinish - uvwMapStart;
    cout << "Transfer uvw Frequency Elapsed Time: " << uvwMapElapsed.count() << " s\n";

    // 开启cpu线程并行
    // 一个线程处理1个GPU
    #pragma omp parallel
    {
        int tid = omp_get_thread_num();  // 从 0 开始编号的并行执行线程
        hipSetDevice(tid);
        std::cout << "Thread " << tid << " is running on device " << tid << std::endl;

        // 遍历所有开启的线程处理， 一个线程控制一个GPU 处理一个id*amount/total的块
        for (int p = tid+start_period; p < days; p += nDevices) 
        {
            cout << "for loop: " << p+1 << endl;

            // 将 l m n C NX 数据从cpu搬到GPU上        
            thrust::device_vector<float> l(cl.begin(), cl.end());
            thrust::device_vector<float> m(cm.begin(), cm.end());
            thrust::device_vector<float> n(cn.begin(), cn.end());
            thrust::device_vector<float> C(cc.begin(), cc.end());

            thrust::device_vector<float> dNX = cNX;
            thrust::device_vector<float> dFF = cFF;
            thrust::device_vector<float> dFF2(NX_index);

            // 创建用来存储不同index中【u, v, w】
            std::vector<float> cu(uvw_presize), cv(uvw_presize), cw(uvw_presize);
            thrust::device_vector<float> u(uvw_presize), v(uvw_presize), w(uvw_presize);

            // 常见用来存储shadeM的 4个 变量
            std::vector<int> M1(uvw_presize), M2(uvw_presize), M3(uvw_presize), M4(uvw_presize);
            thrust::device_vector<int> shadeMat1(uvw_presize), shadeMat2(uvw_presize), shadeMat3(uvw_presize), shadeMat4(uvw_presize);

            // 创建存储uvw坐标对应的频次
            std::vector<float> uvwMapVector(uvw_presize);
            thrust::device_vector<float> uvwFrequencyMap(uvw_presize);
        
            // 存储计算后的到的最终结果
            thrust::device_vector<Complex> F(lmnC_index);

            // 计时统计
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            // 记录开始事件
            hipEventRecord(start);

            // 记录uvw开始事件
            hipEvent_t uvwstart, uvwstop;
            hipEventCreate(&uvwstart);
            hipEventCreate(&uvwstop);
            hipEventRecord(uvwstart);

            // 创建一个临界区，保证只有一个线程进入，用于构建u v w
            int uvw_index, shade_index;
            #pragma omp critical
            {
                // 读取uvw
                string address_uvw = address + "uvw" + to_string(p+1) + duration + sufix;
                cout << "address_uvw: " << address_uvw << std::endl;
                
                ifstream uvwFile(address_uvw);
                // 同时用一个向量保存每一个uvw坐标点的frequency
                uvw_index = 0;
                float u_point, v_point, w_point;
                string key_point;
                if (uvwFile.is_open()) {
                    while (uvwFile >> u_point >> v_point >> w_point) {
                        // 直接构造 key_point
                        key_point = std::to_string(static_cast<int>(u_point)) + "_" + 
                                    std::to_string(static_cast<int>(v_point)) + "_" + 
                                    std::to_string(static_cast<int>(w_point));

                        // 简化查找操作
                        auto it = cUVWFrequencyMap.find(key_point);
                        if (it != cUVWFrequencyMap.end()) {
                            uvwMapVector[uvw_index] = 1 / (it->second);  // 存储频次的倒数
                        } else {
                            uvwMapVector[uvw_index] = 1; 
                        }
                        // cu, cv, cw 需要存储原始坐标
                        cu[uvw_index] = u_point;
                        cv[uvw_index] = v_point;
                        cw[uvw_index] = w_point;
                        uvw_index++;
                    }
                }               
                cout << "load uvw with uvw_index: " << uvw_index << endl; 
                // 复制到GPU上
                thrust::copy(cu.begin(), cu.begin() + uvw_index, u.begin());
                thrust::copy(cv.begin(), cv.begin() + uvw_index, v.begin());
                thrust::copy(cw.begin(), cw.begin() + uvw_index, w.begin());
                thrust::copy(uvwMapVector.begin(), uvwMapVector.begin() + uvw_index, uvwFrequencyMap.begin());

                // 读取shadeM
                string address_shadeM = address + "shadeM" + to_string(p+1) + duration + sufix;
                cout << "address_shadeM: " << address_shadeM << std::endl;
                ifstream shadeMFile(address_shadeM);
                shade_index = 0;
                if (shadeMFile.is_open()) {
                    int shadeM1, shadeM2, shadeM3, shadeM4;
                    // matlab中是从1开始，因此所有值都减去 1
                    while (shadeMFile >> shadeM1 >> shadeM2 >> shadeM3 >> shadeM4) {
                        if(shadeM1 > shadeM2){
                            M1[shade_index] = shadeM2-1;
                            M2[shade_index] = shadeM1-1;
                        }else{
                            M1[shade_index] = shadeM1-1;
                            M2[shade_index] = shadeM2-1;
                        }
                        if(shadeM3 > shadeM4){
                            M3[shade_index] = shadeM4-1;
                            M4[shade_index] = shadeM3-1;
                        }else{
                            M3[shade_index] = shadeM3-1;
                            M4[shade_index] = shadeM4-1;
                        }
                        shade_index++;
                    }
                }
                cout << "load shade matrix with shade_index: " << shade_index << endl; 
                if(shade_index != uvw_index){
                    cout << "load wrong! uvw shape must be equal to shadeM shape" << endl;
                }else{
                    cout << "load right! uvw shape is equal to shadeM shape" << endl;
                }
                // 复制到GPU上
                thrust::copy(M1.begin(), M1.begin() + shade_index, shadeMat1.begin());
                thrust::copy(M2.begin(), M2.begin() + shade_index, shadeMat2.begin());
                thrust::copy(M3.begin(), M3.begin() + shade_index, shadeMat3.begin());
                thrust::copy(M4.begin(), M4.begin() + shade_index, shadeMat4.begin()); 
            }

            // 记录uvw结束事件
            hipEventRecord(uvwstop);
            hipEventSynchronize(uvwstop);
            // 计算经过的时间
            float uvwMS = 0;
            hipEventElapsedTime(&uvwMS, uvwstart, uvwstop);
            printf("Period %d Load UWV Cost Time is: %f s\n", p+1, uvwMS/1000);
            // 销毁事件
            hipEventDestroy(uvwstart);
            hipEventDestroy(uvwstop);


            // 记录viss开始事件
            hipEvent_t vissstart, vissstop;
            hipEventCreate(&vissstart);
            hipEventCreate(&vissstop);
            hipEventRecord(vissstart);

            int blockSize;
            int minGridSize; // 最小网格大小

            // 先提前计算每个 q 值的索引有几个
            thrust::device_vector<int> countLoc(lmnC_index);
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeLocCount, 0, 0);
            int gridSize = floor(lmnC_index + blockSize - 1) / blockSize;
            computeLocCount<<<gridSize, blockSize>>>(
                thrust::raw_pointer_cast(dNX.data()), 
                thrust::raw_pointer_cast(countLoc.data()), 
                lmnC_index, NX_index);
            CHECK(hipDeviceSynchronize());

            // 然后存下来每个 q 值对应的索引
            thrust::device_vector<int> NXq(dNX.size());
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeLocViss, 0, 0);
            gridSize = floor(lmnC_index + blockSize - 1) / blockSize;
            cout << "Compute LocViss, girdSize: " << gridSize << endl;
            cout << "Compute LocViss, blockSize: " << blockSize << endl;
            printf("Compute LocViss... Here is gpu %d running process %d\n", omp_get_thread_num(), p+1);
            computeLocViss<<<gridSize, blockSize>>>(
                thrust::raw_pointer_cast(dNX.data()), 
                thrust::raw_pointer_cast(NXq.data()), 
                thrust::raw_pointer_cast(countLoc.data()), 
                lmnC_index, NX_index);
            CHECK(hipDeviceSynchronize());

            // 存储计算后的可见度
            thrust::device_vector<Complex> viss(uvw_index);
            // 调用函数计算可见度
            launch_visscal(uvw_index, lmnC_index, RES,
                    thrust::raw_pointer_cast(viss.data()),
                    thrust::raw_pointer_cast(dFF.data()),
                    thrust::raw_pointer_cast(u.data()),
                    thrust::raw_pointer_cast(v.data()),
                    thrust::raw_pointer_cast(w.data()),
                    thrust::raw_pointer_cast(l.data()),
                    thrust::raw_pointer_cast(m.data()),
                    thrust::raw_pointer_cast(n.data()),
                    thrust::raw_pointer_cast(shadeMat1.data()),
                    thrust::raw_pointer_cast(shadeMat2.data()), 
                    thrust::raw_pointer_cast(shadeMat3.data()), 
                    thrust::raw_pointer_cast(shadeMat4.data()),
                    thrust::raw_pointer_cast(NXq.data()),
                    thrust::raw_pointer_cast(countLoc.data()),
                    I1, CPI, zero, two, dl, dm, dn);
            cout << "period " << p+1 << " viss compute success!" << endl;

            // 记录viss结束事件
            hipEventRecord(vissstop);
            hipEventSynchronize(vissstop);
            // 计算经过的时间
            float vissMS = 0;
            hipEventElapsedTime(&vissMS, vissstart, vissstop);
            printf("Period %d Compute Viss Cost Time is: %f s\n", p+1, vissMS/1000);
            // 销毁事件
            hipEventDestroy(vissstart);
            hipEventDestroy(vissstop);


            // 记录imagerecon开始事件
            hipEvent_t imagereconstart, imagereconstop;
            hipEventCreate(&imagereconstart);
            hipEventCreate(&imagereconstop);
            hipEventRecord(imagereconstart);

            // 创建预处理的theta 和 phi
            thrust::device_vector<float> thetaP0(uvw_presize), phiP0(uvw_presize), dtheta(uvw_presize), dphi(uvw_presize);
            // 直接在GPU上计算
            // 因为前面加载的时候确保了M1 > M2, M3 > M4, 因此abs函数可以去掉
            thrust::transform(shadeMat1.begin(), shadeMat1.end(), shadeMat2.begin(), thetaP0.begin(), thrust::divides<float>());
            thrust::transform(shadeMat3.begin(), shadeMat3.end(), shadeMat4.begin(), phiP0.begin(), thrust::divides<float>());
            thrust::transform(shadeMat1.begin(), shadeMat1.end(), shadeMat2.begin(), dtheta.begin(), thrust::divides<float>());
            thrust::transform(shadeMat3.begin(), shadeMat3.end(), shadeMat4.begin(), dphi.begin(), thrust::divides<float>());
            
            // 调用image_recon函数计算图像反演
            launch_imagerecon(uvw_index, lmnC_index, RES,
                thrust::raw_pointer_cast(F.data()),
                thrust::raw_pointer_cast(viss.data()),
                thrust::raw_pointer_cast(u.data()),
                thrust::raw_pointer_cast(v.data()),
                thrust::raw_pointer_cast(w.data()),
                thrust::raw_pointer_cast(l.data()),
                thrust::raw_pointer_cast(m.data()),
                thrust::raw_pointer_cast(n.data()),
                thrust::raw_pointer_cast(uvwFrequencyMap.data()),
                thrust::raw_pointer_cast(thetaP0.data()),
                thrust::raw_pointer_cast(phiP0.data()),
                thrust::raw_pointer_cast(dtheta.data()),
                thrust::raw_pointer_cast(dphi.data()),
                I1, CPI, zero, two, dl, dm, dn);
            cout << "Period " << p+1 << " Image Reconstruction Success!" << endl;
            
            // 记录imagerecon结束事件
            hipEventRecord(imagereconstop);
            hipEventSynchronize(imagereconstop);
            // 计算经过的时间
            float imagereconMS = 0;
            hipEventElapsedTime(&imagereconMS, imagereconstart, imagereconstop);
            printf("Period %d Image Reconstruction Cost Time is: %f s\n", p+1, imagereconMS/1000);
            // 销毁事件
            hipEventDestroy(imagereconstart);
            hipEventDestroy(imagereconstop);


            // 记录saveimage开始事件
            hipEvent_t saveimagestart, saveimagestop;
            hipEventCreate(&saveimagestart);
            hipEventCreate(&saveimagestop);
            hipEventRecord(saveimagestart);
            // 创建一个临界区，用于保存图像反演结果
            #pragma omp critical
            {   
                // 在CPU上创建变量保存F结果
                thrust::host_vector<Complex> tempF = F;
                thrust::host_vector<Complex> extendF(NX_index);

                std::ofstream F_File;
                string address_F = "F_recon_1M/F" + to_string(p+1) + "_optim2.txt";
                cout << "address_F: " << address_F << endl;
                F_File.open(address_F);
                if (!F_File.is_open()) {
                    std::cerr << "Error opening file: " << address_F << endl;
                }
                for (int c = 0; c < NX_index; c++) {
                    int tmp = static_cast<int>(cNX[c]) - 1;  // matlab中下标从1开始
                    extendF[c] = tempF[tmp];
                    F_File << extendF[c].real() << std::endl;
                }
                F_File.close();
                std::cout << "Period " << p+1 << " save F success!" << std::endl;
            }

            // 记录saveimage结束事件
            hipEventRecord(saveimagestop);
            hipEventSynchronize(saveimagestop);
            // 计算经过的时间
            float saveimageMS = 0;
            hipEventElapsedTime(&saveimageMS, saveimagestart, saveimagestop);
            printf("Period %d Save Image Cost Time is: %f s\n", p+1, saveimageMS/1000);
            // 销毁事件
            hipEventDestroy(saveimagestart);
            hipEventDestroy(saveimagestop);

            // 记录全程结束事件
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            // 计算经过的时间
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Period %d Elapsed time: %f s\n", p+1, milliseconds/1000);
            // 销毁事件
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }
    
    gettimeofday(&finish, NULL);
    total_time = ((finish.tv_sec - start.tv_sec) * 1000000 + (finish.tv_usec - start.tv_usec)) / 1000000.0;
    cout << "total time: " << total_time << "s" << endl;
    return 0;
}


int main()
{
    int start_period = 0;  // 从哪个周期开始，一共是130个周期
    vissGen(0, 2094, start_period);
}

